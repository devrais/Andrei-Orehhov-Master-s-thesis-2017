
#include <hip/hip_runtime.h>
#include <string.h>

__device__
size_t strlen(const char *str)
{
	const char *s;

	for (s = str; *s; ++s)
		;
	return (s - str);
}

__device__
int strcmp(const char *s1, const char *s2)
{
	while (*s1 == *s2++)
		if (*s1++ == 0)
			return (0);
	return (*(unsigned char *)s1 - *(unsigned char *)--s2);
}

__device__
int memcmp(const void *s1, const void *s2, size_t n)
{
	if (n != 0) {
		const unsigned char *p1 = (const unsigned char *)s1, *p2 = (const unsigned char *)s2;

		do {
			if (*p1++ != *p2++)
				return (*--p1 - *--p2);
		} while (--n != 0);
	}
	return (0);
}
