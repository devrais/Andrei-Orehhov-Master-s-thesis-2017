#include "hip/hip_runtime.h"
/** Find leftmost node containing given value
 *  returns NULL if node was not found
 */
#include <stdio.h>
#include <string.h>
#include <stdlib.h>

#include "../config.h"

#include "dbdata.h"
#include "dbindex.h"
#include "dbcompare.h"
#include "dbhash.h"

/* ====== Private defs =========== */

#define LL_CASE 0
#define LR_CASE 1
#define RL_CASE 2
#define RR_CASE 3

#ifndef max
#define max(a,b) (a>b ? a : b)
#endif

#define HASHIDX_OP_STORE 1
#define HASHIDX_OP_REMOVE 2
#define HASHIDX_OP_FIND 3

#ifndef TTREE_SINGLE_COMPARE

#else
/* "rightmost" node search is the improved tree search described in
 * the original T-tree paper.
 */
#define db_find_bounding_tnode wg_search_ttree_rightmost
#endif

#define INDEX_ADD_ROW(d, h, i, r) \
  switch(h->type) { \
    case WG_INDEX_TYPE_TTREE: \
      if(ttree_add_row(d, i, r)) \
        return -2; \
      break; \
    case WG_INDEX_TYPE_TTREE_JSON: \
      if(is_plain_record(r)) { \
        if(ttree_add_row(d, i, r)) \
          return -2; \
      } \
      break; \
    case WG_INDEX_TYPE_HASH: \
      if(hash_add_row(d, i, r)) \
        return -2; \
      break; \
    case WG_INDEX_TYPE_HASH_JSON: \
      if(is_plain_record(r)) { \
        if(hash_add_row(d, i, r)) \
          return -2; \
      } \
      break; \
    default: \
      show_index_error(db, "unknown index type, ignoring"); \
      break; \
  }

#define INDEX_REMOVE_ROW(d, h, i, r) \
  switch(h->type) { \
    case WG_INDEX_TYPE_TTREE: \
      if(ttree_remove_row(d, i, r) < -2) \
        return -2; \
      break; \
    case WG_INDEX_TYPE_TTREE_JSON: \
      if(is_plain_record(r)) { \
        if(ttree_remove_row(d, i, r) < -2) \
          return -2; \
      } \
      break; \
    case WG_INDEX_TYPE_HASH: \
      if(hash_remove_row(d, i, r) < -2) \
        return -2; \
      break; \
    case WG_INDEX_TYPE_HASH_JSON: \
      if(is_plain_record(r)) { \
        if(hash_remove_row(d, i, r) < -2) \
          return -2; \
      } \
      break; \
    default: \
      show_index_error(db, "unknown index type, ignoring"); \
      break; \
  }

/* ======= Private protos ================ */
__device__  static gint ttree_add_row(void *db, gint index_id, void *rec);
__device__  static gint hash_add_row(void *db, gint index_id, void *rec);
__device__  static gint show_index_error(void* db, char* errmsg);

__device__  static gint hash_recurse(void *db, wg_index_header *hdr,
		char *prefix, gint prefixlen, gint *values, gint count, void *rec,
		gint op, gint expand);
__device__ static gint hash_extend_prefix(void *db, wg_index_header *hdr, char *prefix,
  gint prefixlen, gint nextval, gint *values, gint count, void *rec, gint op,
  gint expand);

__device__ static int db_rotate_ttree(void *db, gint index_id, struct wg_tnode *root,
  int overw);

__device__ static int db_which_branch_causes_overweight(void *db, struct wg_tnode *root);
__device__ static gint ttree_remove_row(void *db, gint index_id, void * rec);

__device__ static gint hash_remove_row(void *db, gint index_id, void *rec);

/* ====== Functions ============== */

__device__ static gint hash_extend_prefix(void *db, wg_index_header *hdr, char *prefix,
  gint prefixlen, gint nextval, gint *values, gint count, void *rec, gint op,
  gint expand) {

  char *fldbytes, *newprefix;
  gint newlen, fldlen, retv;

  fldlen = wg_decode_for_hashing(db, nextval, &fldbytes);
  if(fldlen < 1) {
    show_index_error(db,"Failed to decode a field value for hash");
    return -1;
  }

  if(prefix && prefixlen) {
    newlen = prefixlen + fldlen + 1;
  } else {
    newlen = fldlen;
  }

  newprefix = (char *)malloc(newlen);
  if(!newprefix) {
    free(fldbytes);
    show_index_error(db, "Failed to allocate memory");
    return -1;
  }
  if(prefix) {
    memcpy(newprefix, prefix, prefixlen);
    newprefix[prefixlen] = '\0'; /* XXX: why? double-check this */
  }

  memcpy(newprefix + (newlen - fldlen), fldbytes, fldlen);
  retv = hash_recurse(db, hdr, newprefix,
    newlen, values, count, rec, op, expand);
  free(fldbytes);
  free(newprefix);
  return retv;
}

__device__ static gint hash_recurse(void *db, wg_index_header *hdr, char *prefix,
  gint prefixlen, gint *values, gint count, void *rec, gint op, gint expand) {

  if(count) {
    gint nextvalue = values[0];
    if(expand) {
      /* In case of a JSON/array index, check the value */
      if(wg_get_encoded_type(db, nextvalue) == WG_RECORDTYPE) {
        void *valrec = wg_decode_record(db, nextvalue);

        if(is_schema_array(valrec)) {
          /* expand the array */
          gint i, reclen, retv = 0;
          reclen = wg_get_record_len(db, valrec);
          for(i=0; i<reclen; i++) {
            retv = hash_extend_prefix(db, hdr, prefix, prefixlen,
              wg_get_field(db, valrec, i),
              &values[1], count - 1, rec, op, expand);
            if(retv)
              break;
          }
          return retv; /* This skips adding the array record itself. It's
                        * not useful as we can only hash the offset. */
        }
      }
    }
    /* Regular index. JSON/array index also falls back to this. */
    return hash_extend_prefix(db, hdr, prefix, prefixlen,
      nextvalue, &values[1], count - 1, rec, op, expand);
  }
  else {
    /* No more values, the hash string is complete. Add it to the index */
    if(op == HASHIDX_OP_STORE) {
      return wg_idxhash_store(db, HASHIDX_ARRAYP(hdr),
        prefix, prefixlen, ptrtooffset(db, rec));
    } else if(op == HASHIDX_OP_REMOVE) {
      return wg_idxhash_remove(db, HASHIDX_ARRAYP(hdr),
        prefix, prefixlen, ptrtooffset(db, rec));
    } else {
      /* assume HASHIDX_OP_FIND */
      return wg_idxhash_find(db, HASHIDX_ARRAYP(hdr), prefix, prefixlen);
    }
  }
  return 0; /* pacify the compiler */
}

__device__  static gint hash_add_row(void *db, gint index_id, void *rec) {
	wg_index_header *hdr = (wg_index_header *) offsettoptr(db, index_id);
	gint i;
	gint values[MAX_INDEX_FIELDS];

	for (i = 0; i < hdr->fields; i++) {
		values[i] = wg_get_field(db, rec, hdr->rec_field_index[i]);
	}
	return hash_recurse(db, hdr, NULL, 0, values, hdr->fields, rec,
	HASHIDX_OP_STORE, (hdr->type == WG_INDEX_TYPE_HASH_JSON));
}

__device__  static gint ttree_add_row(void *db, gint index_id, void *rec) {
	gint rootoffset, column;
	gint newvalue, boundtype, bnodeoffset, newoffset;
	struct wg_tnode *node;
	wg_index_header *hdr = (wg_index_header *) offsettoptr(db, index_id);
	db_memsegment_header* dbh = dbmemsegh(db);

	rootoffset = TTREE_ROOT_NODE(hdr);
#ifdef CHECK
	if (rootoffset == 0) {
#ifdef WG_NO_ERRPRINT
#else
		//printf("index at offset %d does not exist\n", (int) index_id);
#endif
		return -1;
	}
#endif
	column = hdr->rec_field_index[0]; /* always one column for T-tree */

	//extract real value from the row (rec)
	newvalue = wg_get_field(db, rec, column);

	//find bounding node for the value
	bnodeoffset = db_find_bounding_tnode(db, rootoffset, newvalue, &boundtype,
			NULL);
	node = (struct wg_tnode *) offsettoptr(db, bnodeoffset);
	newoffset = 0; //save here the offset of newly created tnode - 0 if no node added into the tree
	//if bounding node exists - follow one algorithm, else the other
	if (boundtype == REALLY_BOUNDING_NODE) {

		//check if the node has room for a new entry
		if (node->number_of_elements < WG_TNODE_ARRAY_SIZE) {
			int i, j;
			gint cr;

			/* add array entry and update control data. We keep the
			 * array sorted, smallest values left. */
			for (i = 0; i < node->number_of_elements; i++) {
				/* The node is small enough for naive scans to be
				 * "good enough" inside the node. Note that we
				 * branch into re-sort loop as early as possible
				 * with >= operator (> would be algorithmically correct too)
				 * since here the compare is more expensive than the slot
				 * copying.
				 */
				cr =
						WG_COMPARE(db,
								wg_get_field(db, (void *)offsettoptr(db,node->array_of_values[i]), column),
								newvalue);

				if (cr != WG_LESSTHAN) { /* value >= newvalue */
					/* Push remaining values to the right */
					for (j = node->number_of_elements; j > i; j--)
						node->array_of_values[j] = node->array_of_values[j - 1];
					break;
				}
			}
			/* i is either number_of_elements or a vacated slot
			 * in the array now. */
			node->array_of_values[i] = ptrtooffset(db, rec);
			node->number_of_elements++;

			/* Update min. Due to the >= comparison max is preserved
			 * in this case. Note that we are overwriting values that
			 * WG_COMPARE() may deem equal. This is intentional, because other
			 * parts of T-tree algorithm rely on encoded values of min/max fields
			 * to be in sync with the leftmost/rightmost slots.
			 */
			if (i == 0) {
				node->current_min = newvalue;
			}
		} else {
			//still, insert the value here, but move minimum out of this node
			//get the minimum element from this node
			int i, j;
			gint cr, minvalue, minvaluerowoffset;

			minvalue = node->current_min;
			minvaluerowoffset = node->array_of_values[0];

			/* Now scan for the matching slot. However, since
			 * we already know the 0 slot will be re-filled, we
			 * do this scan (and sort) in reverse order, compared to the case
			 * where array had some space left. */
			for (i = WG_TNODE_ARRAY_SIZE - 1; i > 0; i--) {
				cr =
						WG_COMPARE(db,
								wg_get_field(db, (void *)offsettoptr(db,node->array_of_values[i]), column),
								newvalue);
				if (cr != WG_GREATER) { /* value <= newvalue */
					/* Push remaining values to the left */
					for (j = 0; j < i; j++)
						node->array_of_values[j] = node->array_of_values[j + 1];
					break;
				}
			}
			/* i is either 0 or a freshly vacated slot */
			node->array_of_values[i] = ptrtooffset(db, rec);

			/* Update minimum. Thanks to the sorted array, we know for a fact
			 * that the minimum sits in slot 0. */
			if (i == 0) {
				node->current_min = newvalue;
			} else {
				node->current_min = wg_get_field(db,
						(void *) offsettoptr(db, node->array_of_values[0]),
						column);
				/* The scan for the free slot starts from the right and
				 * tries to exit as fast as possible. So it's possible that
				 * the rightmost slot was changed.
				 */
				if (i == WG_TNODE_ARRAY_SIZE - 1) {
					node->current_max = newvalue;
				}
			}

			//proceed to the node that holds greatest lower bound - must be leaf (can be the initial bounding node)
			if (node->left_child_offset != 0) {
#ifndef TTREE_CHAINED_NODES
				gint greatestlb = wg_ttree_find_glb_node(db,node->left_child_offset);
#else
				gint greatestlb = node->pred_offset;
#endif
				node = (struct wg_tnode *) offsettoptr(db, greatestlb);
			}
			//if the greatest lower bound node has room, insert value
			//otherwise make the new node as right child and put the value there
			if (node->number_of_elements < WG_TNODE_ARRAY_SIZE) {
				//add array entry and update control data
				node->array_of_values[node->number_of_elements] =
						minvaluerowoffset;    //save offset, use first free slot
				node->number_of_elements++;
				node->current_max = minvalue;

			} else {
				//create, initialize and save first value
				struct wg_tnode *leaf;
				gint newnode = wg_alloc_fixlen_object(db,
						&dbh->tnode_area_header);
				if (newnode == 0)
					return -1;
				leaf = (struct wg_tnode *) offsettoptr(db, newnode);
				leaf->parent_offset = ptrtooffset(db, node);
				leaf->left_subtree_height = 0;
				leaf->right_subtree_height = 0;
				leaf->current_max = minvalue;
				leaf->current_min = minvalue;
				leaf->number_of_elements = 1;
				leaf->left_child_offset = 0;
				leaf->right_child_offset = 0;
				leaf->array_of_values[0] = minvaluerowoffset;
				/* If the original, full node did not have a left child, then
				 * there also wasn't a separate GLB node, so we are adding one now
				 * as the left child. Otherwise, the new node is added as the right
				 * child to the current GLB node.
				 */
				if (bnodeoffset == ptrtooffset(db, node)) {
					node->left_child_offset = newnode;
#ifdef TTREE_CHAINED_NODES
					/* Create successor / predecessor relationship */
					leaf->succ_offset = ptrtooffset(db, node);
					leaf->pred_offset = node->pred_offset;

					if (node->pred_offset) {
						struct wg_tnode *pred = (struct wg_tnode *) offsettoptr(
								db, node->pred_offset);
						pred->succ_offset = newnode;
					} else {
						TTREE_MIN_NODE(hdr) = newnode;
					}
					node->pred_offset = newnode;
#endif
				} else {
#ifdef TTREE_CHAINED_NODES
					struct wg_tnode *succ;
#endif
					node->right_child_offset = newnode;
#ifdef TTREE_CHAINED_NODES
					/* Insert the new node in the sequential chain between
					 * the original node and the GLB node found */
					leaf->succ_offset = node->succ_offset;
					leaf->pred_offset = ptrtooffset(db, node);

#ifdef CHECK
					if (!node->succ_offset) {
						show_index_error(db, "GLB with no successor, panic");
						return -1;
					} else {
#endif
						succ = (struct wg_tnode *) offsettoptr(db,
								leaf->succ_offset);
						succ->pred_offset = newnode;
#ifdef CHECK
					}
#endif
					node->succ_offset = newnode;
#endif /* TTREE_CHAINED_NODES */
				}
				newoffset = newnode;
			}
		}

	}      //the bounding node existed - first algorithm
	else {      // bounding node does not exist
				//try to insert the new value to that node - becoming new min or max
				//if the node has room for a new entry
		if (node->number_of_elements < WG_TNODE_ARRAY_SIZE) {
			int i;

			/* add entry, keeping the array sorted (see also notes for the
			 * bounding node case. The difference this time is that we already
			 * know if this value is becoming the new min or max).
			 */
			if (boundtype == DEAD_END_LEFT_NOT_BOUNDING) {
				/* our new value is the new min, push everything right */
				for (i = node->number_of_elements; i > 0; i--)
					node->array_of_values[i] = node->array_of_values[i - 1];
				node->array_of_values[0] = ptrtooffset(db, rec);
				node->current_min = newvalue;
			} else { /* DEAD_END_RIGHT_NOT_BOUNDING */
				/* even simpler case, new value is added to the right */
				node->array_of_values[node->number_of_elements] = ptrtooffset(
						db, rec);
				node->current_max = newvalue;
			}

			node->number_of_elements++;

			/* XXX: not clear if the empty node can occur here. Until this
			 * is checked, we'll be paranoid and overwrite both min and max. */
			if (node->number_of_elements == 1) {
				node->current_max = newvalue;
				node->current_min = newvalue;
			}
		} else {
			//make a new node and put data there
			struct wg_tnode *leaf;
			gint newnode = wg_alloc_fixlen_object(db, &dbh->tnode_area_header);
			if (newnode == 0)
				return -1;
			leaf = (struct wg_tnode *) offsettoptr(db, newnode);
			leaf->parent_offset = ptrtooffset(db, node);
			leaf->left_subtree_height = 0;
			leaf->right_subtree_height = 0;
			leaf->current_max = newvalue;
			leaf->current_min = newvalue;
			leaf->number_of_elements = 1;
			leaf->left_child_offset = 0;
			leaf->right_child_offset = 0;
			leaf->array_of_values[0] = ptrtooffset(db, rec);
			newoffset = newnode;
			//set new node as left or right leaf
			if (boundtype == DEAD_END_LEFT_NOT_BOUNDING) {
				node->left_child_offset = newnode;
#ifdef TTREE_CHAINED_NODES
				/* Set the new node as predecessor of the parent */
				leaf->succ_offset = ptrtooffset(db, node);
				leaf->pred_offset = node->pred_offset;

				if (node->pred_offset) {
					/* Notify old predecessor that the node following
					 * it changed */
					struct wg_tnode *pred = (struct wg_tnode *) offsettoptr(db,
							node->pred_offset);
					pred->succ_offset = newnode;
				} else {
					TTREE_MIN_NODE(hdr) = newnode;
				}
				node->pred_offset = newnode;
#endif
			} else if (boundtype == DEAD_END_RIGHT_NOT_BOUNDING) {
				node->right_child_offset = newnode;
#ifdef TTREE_CHAINED_NODES
				/* Set the new node as successor of the parent */
				leaf->succ_offset = node->succ_offset;
				leaf->pred_offset = ptrtooffset(db, node);

				if (node->succ_offset) {
					/* Notify old successor that the node preceding
					 * it changed */
					struct wg_tnode *succ = (struct wg_tnode *) offsettoptr(db,
							node->succ_offset);
					succ->pred_offset = newnode;
				} else {
					TTREE_MAX_NODE(hdr) = newnode;
				}
				node->succ_offset = newnode;
#endif
			}
		}
	}    //no bounding node found - algorithm 2

	//if new node was added to tree - must update child height data in nodes from leaf to root
	//or until find a node with imbalance
	//then determine the bad balance case: LL, LR, RR or RL and execute proper rotation
	if (newoffset) {
		struct wg_tnode *child = (struct wg_tnode *) offsettoptr(db, newoffset);
		struct wg_tnode *parent;
		int left = 0;
		while (child->parent_offset != 0) {  //this is not a root
			int balance;
			parent = (struct wg_tnode *) offsettoptr(db, child->parent_offset);
			//determine which child the child is, left or right one
			if (parent->left_child_offset == ptrtooffset(db, child))
				left = 1;
			else
				left = 0;
			//increment parent left or right subtree height
			if (left)
				parent->left_subtree_height++;
			else
				parent->right_subtree_height++;

			//check balance
			balance = parent->left_subtree_height
					- parent->right_subtree_height;
			if (balance == 0) {
				/* As a result of adding a new node somewhere below, left
				 * and right subtrees of the node we're checking became
				 * of EQUAL height. This means that changes in subtree heights
				 * do not propagate any further (the max depth in this node
				 * dit NOT change).
				 */
				break;
			}
			if (balance > 1 || balance < -1) {  //must rebalance
			//the current parent is root for balancing operation
			//determine the branch that causes overweight
				int overw = db_which_branch_causes_overweight(db, parent);
				//fix balance
				db_rotate_ttree(db, index_id, parent, overw);
				break; //while loop because balance does not change in the next levels
			} else {        //just proceed to the parent node
				child = parent;
			}
		}
	}
	return 0;
}

__device__ gint wg_index_add_rec(void *db, void *rec) {
	gint i;
	db_memsegment_header* dbh = dbmemsegh(db);
	gint reclen = wg_get_record_len(db, rec);

#ifdef CHECK
	if (is_special_record(rec))
		return -1;
#endif

	if (reclen > MAX_INDEXED_FIELDNR)
		reclen = MAX_INDEXED_FIELDNR + 1;

	for (i = 0; i < reclen; i++) {
		gint *ilist;
		gcell *ilistelem;

		/* Find all indexes on the column */
		ilist = &dbh->index_control_area_header.index_table[i];
		while (*ilist) {
			ilistelem = (gcell *) offsettoptr(db, *ilist);
			if (ilistelem->car) {
				wg_index_header *hdr = (wg_index_header *) offsettoptr(db,
						ilistelem->car);
				if (hdr->rec_field_index[hdr->fields - 1] == i) {
					/* Only add the record if we're at the last column
					 * of the index. This way we ensure that a.) a record
					 * is entered once into a multi-column index and b.) the
					 * record is long enough so that it qualifies for the
					 * multi-column index.
					 * For a single-column index, the indexed column is
					 * also the last column, therefore the above is valid,
					 * altough the check is unnecessary.
					 */
					if (MATCH_TEMPLATE(db, hdr, rec)) {
						INDEX_ADD_ROW(db, hdr, ilistelem->car, rec)
					}
				}
			}
			ilist = &ilistelem->cdr;
		}

#ifdef USE_INDEX_TEMPLATE
		ilist = &dbh->index_control_area_header.index_template_table[i];
		while (*ilist) {
			ilistelem = (gcell *) offsettoptr(db, *ilist);
			if (ilistelem->car) {
				wg_index_header *hdr = (wg_index_header *) offsettoptr(db,
						ilistelem->car);
				wg_index_template *tmpl = (wg_index_template *) offsettoptr(db,
						hdr->template_offset);
				void *matchrec;
				gint mreclen;
				int j, firstmatch = -1;

				/* Here the check for a match is slightly more complicated.
				 * If there is a match *but* the current column is not the
				 * first fixed one in the template, the match has
				 * already occurred earlier.
				 */
				matchrec = offsettoptr(db, tmpl->offset_matchrec);
				mreclen = wg_get_record_len(db, matchrec);
				if (mreclen > reclen) {
					goto nexttmpl1;
				}
				for (j = 0; j < mreclen; j++) {
					gint enc = wg_get_field(db, matchrec, j);
					if (wg_get_encoded_type(db, enc) != WG_VARTYPE) {
						if (WG_COMPARE(db, enc,
								wg_get_field(db, rec, j)) != WG_EQUAL)
							goto nexttmpl1;
						if (firstmatch < 0)
							firstmatch = j;
					}
				}
				if (firstmatch == i
						&& reclen > hdr->rec_field_index[hdr->fields - 1]) {
					/* The record matches AND this is the first time we
					 * see this index. Update it.
					 */
					INDEX_ADD_ROW(db, hdr, ilistelem->car, rec)
				}
			}
			nexttmpl1: ilist = &ilistelem->cdr;
		}
#endif

	}
	return 0;
}

#ifdef USE_INDEX_TEMPLATE
__device__
   static gint find_index_template(void *db, gint *matchrec,
		gint reclen);
#endif

__device__
   static gint sort_columns(gint *sorted_cols, gint *columns,
		gint col_count);
__device__
 static gint show_index_error(void* db, char* errmsg);
__device__
 static gint show_index_error_nr(void* db, char* errmsg, gint nr);

__device__ gint wg_search_ttree_rightmost(void *db, gint rootoffset,  // 1180
		gint key, gint *result, struct wg_tnode *rb_node) {

	struct wg_tnode * node;

#ifdef TTREE_SINGLE_COMPARE
	node = (struct wg_tnode *) offsettoptr(db, rootoffset);

	/* Improved(?) tree search algorithm with a single compare per node.
	 * only lower bound is examined, if the value is larger the right subtree
	 * is selected immediately. If the search ends in a dead end, the node where
	 * the right branch was taken is examined again.
	 */
	if (WG_COMPARE(db, key, node->current_min) == WG_LESSTHAN) {
		/* key < node->current_min */
		if (node->left_child_offset != 0) {
			return wg_search_ttree_rightmost(db, node->left_child_offset, key,
					result, rb_node);
		} else if (rb_node) {
			/* Dead end, but we still have an unexamined node left */
			if (WG_COMPARE(db, key, rb_node->current_max) != WG_GREATER) {
				/* key<=rb_node->current_max */
				*result = REALLY_BOUNDING_NODE;
				return ptrtooffset(db, rb_node);
			}
		}
		/* No left child, no rb_node or it's right bound was not interesting */
		*result = DEAD_END_LEFT_NOT_BOUNDING;
		return rootoffset;
	} else {
		if (node->right_child_offset != 0) {
			/* Here we jump the gun and branch to right, ignoring the
			 * current_max of the node (therefore avoiding one expensive
			 * compare operation).
			 */
			return wg_search_ttree_rightmost(db, node->right_child_offset, key,
					result, node);
		} else if (WG_COMPARE(db, key, node->current_max) != WG_GREATER) {
			/* key<=node->current_max */
			*result = REALLY_BOUNDING_NODE;
			return rootoffset;
		}
		/* key is neither left of or inside this node and
		 * there is no right child */
		*result = DEAD_END_RIGHT_NOT_BOUNDING;
		return rootoffset;
	}
#else
	gint bnodeoffset;

	bnodeoffset = db_find_bounding_tnode(db, rootoffset, key, result, NULL);
	if(*result != REALLY_BOUNDING_NODE)
	return bnodeoffset;

	/* There is at least one node with the key we're interested in,
	 * now make sure we have the rightmost */
	node = offsettoptr(db, bnodeoffset);
	while(WG_COMPARE(db, node->current_max, key) == WG_EQUAL) {
		gint nextoffset = TNODE_SUCCESSOR(db, node);
		if(nextoffset) {
			struct wg_tnode *next = offsettoptr(db, nextoffset);
			if(WG_COMPARE(db, next->current_min, key) == WG_GREATER)
			/* next->current_min > key */
			break; /* overshot */
			node = next;
		}
		else
		break; /* last node in chain */
	}
	return ptrtooffset(db, node);
#endif
}

__device__ gint wg_search_ttree_leftmost(void *db, gint rootoffset,  //1257
		gint key, gint *result, struct wg_tnode *lb_node) {

	struct wg_tnode * node;

#ifdef TTREE_SINGLE_COMPARE
	node = (struct wg_tnode *) offsettoptr(db, rootoffset);

	/* Rightmost bound search mirrored */
	if (WG_COMPARE(db, key, node->current_max) == WG_GREATER) {
		/* key > node->current_max */
		if (node->right_child_offset != 0) {
			return wg_search_ttree_leftmost(db, node->right_child_offset, key,
					result, lb_node);
		} else if (lb_node) {
			/* Dead end, but we still have an unexamined node left */
			if (WG_COMPARE(db, key, lb_node->current_min) != WG_LESSTHAN) {
				/* key>=lb_node->current_min */
				*result = REALLY_BOUNDING_NODE;
				return ptrtooffset(db, lb_node);
			}
		}
		*result = DEAD_END_RIGHT_NOT_BOUNDING;
		return rootoffset;
	} else {
		if (node->left_child_offset != 0) {
			return wg_search_ttree_leftmost(db, node->left_child_offset, key,
					result, node);
		} else if (WG_COMPARE(db, key, node->current_min) != WG_LESSTHAN) {
			/* key>=node->current_min */
			*result = REALLY_BOUNDING_NODE;
			return rootoffset;
		}
		*result = DEAD_END_LEFT_NOT_BOUNDING;
		return rootoffset;
	}
#else
	gint bnodeoffset;

	bnodeoffset = db_find_bounding_tnode(db, rootoffset, key, result, NULL);
	if(*result != REALLY_BOUNDING_NODE)
	return bnodeoffset;

	/* One (we don't know which) bounding node found, traverse the
	 * tree to the leftmost. */
	node = offsettoptr(db, bnodeoffset);
	while(WG_COMPARE(db, node->current_min, key) == WG_EQUAL) {
		gint prevoffset = TNODE_PREDECESSOR(db, node);
		if(prevoffset) {
			struct wg_tnode *prev = offsettoptr(db, prevoffset);
			if(WG_COMPARE(db, prev->current_max, key) == WG_LESSTHAN)
			/* prev->current_max < key */
			break; /* overshot */
			node = prev;
		}
		else
		break; /* first node in chain */
	}
	return ptrtooffset(db, node);
#endif
}

/** Find first occurrence of a value in a T-tree node
 *  returns the number of the slot. If the value itself
 *  is missing, the location of the first value that
 *  exceeds it is returned.
 */
__device__ gint wg_search_tnode_first(void *db, gint nodeoffset, gint key, // 1325
		gint column) {

	gint i, encoded;
	struct wg_tnode *node = (struct wg_tnode *) offsettoptr(db, nodeoffset);

	for (i = 0; i < node->number_of_elements; i++) {
		/* Naive scan is ok for small values of WG_TNODE_ARRAY_SIZE. */
		encoded = wg_get_field(db,
				(void *) offsettoptr(db, node->array_of_values[i]), column);
		if (WG_COMPARE(db, encoded, key) != WG_LESSTHAN)
			/* encoded >= key */
			return i;
	}

	return -1;
}

__device__ gint wg_search_tnode_last(void *db, gint nodeoffset, gint key, // 1348
		gint column) {

	gint i, encoded;
	struct wg_tnode *node = (struct wg_tnode *) offsettoptr(db, nodeoffset);

	for (i = node->number_of_elements - 1; i >= 0; i--) {
		encoded = wg_get_field(db,
				(void *) offsettoptr(db, node->array_of_values[i]), column);
		if (WG_COMPARE(db, encoded, key) != WG_GREATER)
			/* encoded <= key */
			return i;
	}

	return -1;
}

__device__ gint wg_index_del_field(void *db, void *rec, gint column) {
	gint *ilist;
	gcell *ilistelem;
	db_memsegment_header* dbh = dbmemsegh(db);
	gint reclen = wg_get_record_len(db, rec);

#ifdef CHECK
	/* XXX: if used from wg_set_field() only, this is redundant */
	if (column > MAX_INDEXED_FIELDNR || column >= reclen)
		return -1;
	if (is_special_record(rec))
		return -1;
#endif

#if 0
	/* XXX: if used from wg_set_field() only, this is redundant */
	if(!dbh->index_control_area_header.index_table[column])
	return -1;
#endif

	/* Find all indexes on the column */
	ilist = &dbh->index_control_area_header.index_table[column];
	while (*ilist) {
		ilistelem = (gcell *) offsettoptr(db, *ilist);
		if (ilistelem->car) {
			wg_index_header *hdr = (wg_index_header *) offsettoptr(db,
					ilistelem->car);

			if (reclen > hdr->rec_field_index[hdr->fields - 1]) {
				if (MATCH_TEMPLATE(db, hdr, rec)) {
					INDEX_REMOVE_ROW(db, hdr, ilistelem->car, rec)
				}
			}
		}
		ilist = &ilistelem->cdr;
	}

#ifdef USE_INDEX_TEMPLATE
	/* Find all indexes on the column */
	ilist = &dbh->index_control_area_header.index_template_table[column];
	while (*ilist) {
		ilistelem = (gcell *) offsettoptr(db, *ilist);
		if (ilistelem->car) {
			wg_index_header *hdr = (wg_index_header *) offsettoptr(db,
					ilistelem->car);

			if (reclen > hdr->rec_field_index[hdr->fields - 1]) {
				if (MATCH_TEMPLATE(db, hdr, rec)) {
					INDEX_REMOVE_ROW(db, hdr, ilistelem->car, rec)
				}
			}
		}
		ilist = &ilistelem->cdr;
	}
#endif

	return 0;
}

__device__
   static gint find_index_template(void *db, gint *matchrec,
		gint reclen) {  // 1858
	gint *ilist;
	void *rec;
	db_memsegment_header* dbh = dbmemsegh(db);
	wg_index_template *tmpl;
	gint fixed_columns = 0, last_fixed = 0;
	int i;

	/* Get some statistics about the match record and validate it */
	for (i = 0; i < reclen; i++) {
		gint type = wg_get_encoded_type(db, matchrec[i]);
		if (type == WG_RECORDTYPE) {
			show_index_error(db, "record links not allowed in index templates");
			return 0;
		}
		if (type != WG_VARTYPE) {
			fixed_columns++;
			last_fixed = i;
		}
	}
	if (!fixed_columns) {
		show_index_error(db, "not a legal match record");
		return 0;
	}
	reclen = last_fixed + 1;

	/* Find a matching template. */
	ilist = &dbh->index_control_area_header.index_template_list;
	while (*ilist) {
		gcell *ilistelem = (gcell *) offsettoptr(db, *ilist);
		if (!ilistelem->car) {
			show_index_error(db, "Invalid header in index tempate list");
			return 0;
		}
		tmpl = (wg_index_template *) offsettoptr(db, ilistelem->car);
		if (tmpl->fixed_columns == fixed_columns) {
			rec = offsettoptr(db, tmpl->offset_matchrec);
			if (reclen != wg_get_record_len(db, rec))
				goto nextelem;
			/* match not possible */
			for (i = 0; i < reclen; i++) {
				if (wg_get_encoded_type(db, matchrec[i]) != WG_VARTYPE) {
					if (WG_COMPARE(db,
							matchrec[i], wg_get_field(db, rec, i)) != WG_EQUAL)
						goto nextelem;
				}
			}
			/* We have a match. */
			return ilistelem->car;
		} else if (tmpl->fixed_columns < fixed_columns) {
			/* No matching record found. New template should be inserted
			 * ahead of current element. */
			break;
		}
		nextelem: ilist = &ilistelem->cdr;
	}

	return 0;
}

__device__
   static gint sort_columns(gint *sorted_cols, gint *columns,  // 2001
		gint col_count) {
	gint i = 0;
	gint prev = -1;
	while (i < col_count) {
		gint lowest = MAX_INDEXED_FIELDNR + 1;
		gint j;
		for (j = 0; j < col_count; j++) {
			if (columns[j] < lowest && columns[j] > prev)
				lowest = columns[j];
		}
		if (lowest == MAX_INDEXED_FIELDNR + 1)
			break;
		sorted_cols[i++] = lowest;
		prev = lowest;
	};
	return i;
}

__device__ gint wg_multi_column_to_index_id(void *db, gint *columns,
		gint col_count, // 2396
		gint type, gint *matchrec, gint reclen) {
	int i;
	gint template_offset = 0;
	db_memsegment_header* dbh = dbmemsegh(db);
	gint *ilist;
	gcell *ilistelem;
	gint sorted_cols[MAX_INDEX_FIELDS];

#ifdef USE_INDEX_TEMPLATE
	/* Validate the match record and find the template */
	if (matchrec) {
		if (!reclen) {
			show_index_error(db, "Zero-length match record not allowed");
			return -1;
		}

		if (reclen > MAX_INDEXED_FIELDNR + 1) {
			show_index_error_nr(db, "Match record too long, max",
			MAX_INDEXED_FIELDNR + 1);
			return -1;
		}

		template_offset = find_index_template(db, matchrec, reclen);
		if (!template_offset) {
			/* No matching template */
			return -1;
		}
	}
#endif

	/* Column count validation */
	if (col_count < 1) {
		show_index_error(db, "need at least one indexed column");
		return -1;
	} else if (col_count > MAX_INDEX_FIELDS) {
		show_index_error_nr(db, "Max allowed indexed fields",
		MAX_INDEX_FIELDS);
		return -1;
	}

	if (col_count > 1) {
		if (sort_columns(sorted_cols, columns, col_count) < col_count) {
			show_index_error(db, "Duplicate columns not allowed");
			return -1;
		}
	} else {
		sorted_cols[0] = columns[0];
	}

	for (i = 0; i < col_count; i++) {
		if (sorted_cols[i] > MAX_INDEXED_FIELDNR) {
			show_index_error_nr(db, "Max allowed column number",
			MAX_INDEXED_FIELDNR);
			return -1;
		}
	}

	/* Find all indexes on the first column */
	ilist = &dbh->index_control_area_header.index_table[sorted_cols[0]];
	while (*ilist) {
		ilistelem = (gcell *) offsettoptr(db, *ilist);
		if (ilistelem->car) {
			wg_index_header *hdr = (wg_index_header *) offsettoptr(db,
					ilistelem->car);
#ifndef USE_INDEX_TEMPLATE
			if(!type || type==hdr->type) {
#else
			if ((!type || type == hdr->type)
					&& hdr->template_offset == template_offset) {
#endif
				if (hdr->fields == col_count) {
					for (i = 0; i < col_count; i++) {
						if (hdr->rec_field_index[i] != sorted_cols[i])
							goto nextindex;
					}
					return ilistelem->car; /* index id */
				}
			}
		}
		nextindex: ilist = &ilistelem->cdr;
	}

	return -1;
}

__device__ gint wg_index_add_field(void *db, void *rec, gint column) {
	gint *ilist;
	gcell *ilistelem;
	db_memsegment_header* dbh = dbmemsegh(db);
	gint reclen = wg_get_record_len(db, rec);

#ifdef CHECK
	/* XXX: if used from wg_set_field() only, this is redundant */
	if (column > MAX_INDEXED_FIELDNR || column >= reclen)
		return -1;
	if (is_special_record(rec))
		return -1;
#endif

#if 0
	/* XXX: if used from wg_set_field() only, this is redundant */
	if(!dbh->index_control_area_header.index_table[column])
	return -1;
#endif

	ilist = &dbh->index_control_area_header.index_table[column];
	while (*ilist) {
		ilistelem = (gcell *) offsettoptr(db, *ilist);
		if (ilistelem->car) {
			wg_index_header *hdr = (wg_index_header *) offsettoptr(db,
					ilistelem->car);
			if (reclen > hdr->rec_field_index[hdr->fields - 1]) {
				if (MATCH_TEMPLATE(db, hdr, rec)) {
					INDEX_ADD_ROW(db, hdr, ilistelem->car, rec)
				}
			}
		}
		ilist = &ilistelem->cdr;
	}

#ifdef USE_INDEX_TEMPLATE
	/* Other candidates are indexes that have match
	 * records. The current record may have become compatible
	 * with their template.
	 */
	ilist = &dbh->index_control_area_header.index_template_table[column];
	while (*ilist) {
		ilistelem = (gcell *) offsettoptr(db, *ilist);
		if (ilistelem->car) {
			wg_index_header *hdr = (wg_index_header *) offsettoptr(db,
					ilistelem->car);
			if (reclen > hdr->rec_field_index[hdr->fields - 1]) {
				if (MATCH_TEMPLATE(db, hdr, rec)) {
					INDEX_ADD_ROW(db, hdr, ilistelem->car, rec)
				}
			}
		}
		ilist = &ilistelem->cdr;
	}
#endif

	return 0;
}

__device__
gint wg_match_template(void *db, wg_index_template *tmpl, void *rec) {
  void *matchrec;
  gint reclen, mreclen;
  int i;

#ifdef CHECK
  /* Paranoia */
  if(!tmpl->offset_matchrec) {
    show_index_error(db, "Invalid match record template");
    return 0;
  }
#endif

  matchrec = offsettoptr(db, tmpl->offset_matchrec);
  mreclen = wg_get_record_len(db, matchrec);
  reclen = wg_get_record_len(db, rec);
  if(mreclen > reclen) {
    /* Match records always end in a fixed column, so
     * this is guaranteed to be a mismatch
     */
    return 0;
  }
  else if(mreclen < reclen) {
    /* Fields outside the template always match */
    reclen = mreclen;
  }
  for(i=0; i<reclen; i++) {
    gint enc = wg_get_field(db, matchrec, i);
    if(wg_get_encoded_type(db, enc) != WG_VARTYPE) {
      if(WG_COMPARE(db, enc, wg_get_field(db, rec, i)) != WG_EQUAL)
        return 0;
    }
  }
  return 1;
}

__device__ static int db_which_branch_causes_overweight(void *db, struct wg_tnode *root){
  struct wg_tnode *child;
  if(root->left_subtree_height > root->right_subtree_height){
    child = (struct wg_tnode *)offsettoptr(db,root->left_child_offset);
    if(child->left_subtree_height >= child->right_subtree_height)return LL_CASE;
    else return LR_CASE;
  }else{
    child = (struct wg_tnode *)offsettoptr(db,root->right_child_offset);
    if(child->left_subtree_height > child->right_subtree_height)return RL_CASE;
    else return RR_CASE;
  }
}

__device__ static int db_rotate_ttree(void *db, gint index_id, struct wg_tnode *root, int overw){
  gint grandparent = root->parent_offset;
  gint initialrootoffset = ptrtooffset(db,root);
  struct wg_tnode *r = NULL;
  struct wg_tnode *g = (struct wg_tnode *)offsettoptr(db,grandparent);
  wg_index_header *hdr = (wg_index_header *)offsettoptr(db,index_id);
  gint column = hdr->rec_field_index[0]; /* always one column for T-tree */

  if(overw == LL_CASE){

/*                       A                          B
*                     B     C                    D     A
*                   D  E             ->        N     E  C
*                  N
*/
    //printf("LL_CASE\n");
    //save some stuff into variables for later use
    gint offset_left_child = root->left_child_offset;
    gint offset_right_grandchild = ((struct wg_tnode *)offsettoptr(db,offset_left_child))->right_child_offset;
    gint right_grandchild_height = ((struct wg_tnode *)offsettoptr(db,offset_left_child))->right_subtree_height;


    //first switch: E goes to A's left child
    root->left_child_offset = offset_right_grandchild;
    root->left_subtree_height = right_grandchild_height;
    if(offset_right_grandchild != 0){
      ((struct wg_tnode *)offsettoptr(db,offset_right_grandchild))->parent_offset = ptrtooffset(db,root);
    }
    //second switch: A goes to B's right child
    ((struct wg_tnode *)offsettoptr(db,offset_left_child)) -> right_child_offset = ptrtooffset(db,root);
    ((struct wg_tnode *)offsettoptr(db,offset_left_child)) -> right_subtree_height = max(root->left_subtree_height,root->right_subtree_height)+1;
    root->parent_offset = offset_left_child;
    //for later grandparent fix
    r = (struct wg_tnode *)offsettoptr(db,offset_left_child);

  }else if(overw == RR_CASE){

/*                       A                          C
*                     B     C                    A     E
*                         D   E         ->     B  D      N
*                              N
*/
    //printf("RR_CASE\n");
    //save some stuff into variables for later use
    gint offset_right_child = root->right_child_offset;
    gint offset_left_grandchild = ((struct wg_tnode *)offsettoptr(db,offset_right_child))->left_child_offset;
    gint left_grandchild_height = ((struct wg_tnode *)offsettoptr(db,offset_right_child))->left_subtree_height;
    //first switch: D goes to A's right child
    root->right_child_offset = offset_left_grandchild;
    root->right_subtree_height = left_grandchild_height;
    if(offset_left_grandchild != 0){
      ((struct wg_tnode *)offsettoptr(db,offset_left_grandchild))->parent_offset = ptrtooffset(db,root);
    }
    //second switch: A goes to C's left child
    ((struct wg_tnode *)offsettoptr(db,offset_right_child)) -> left_child_offset = ptrtooffset(db,root);
    ((struct wg_tnode *)offsettoptr(db,offset_right_child)) -> left_subtree_height = max(root->right_subtree_height,root->left_subtree_height)+1;
    root->parent_offset = offset_right_child;
    //for later grandparent fix
    r = (struct wg_tnode *)offsettoptr(db,offset_right_child);

  }else if(overw == LR_CASE){
/*               A                    E
*             B     C             B       A
*          D    E        ->     D  F    G    C
*             F  G                 N
*             N
*/
    struct wg_tnode *bb, *ee;
    //save some stuff into variables for later use
    gint offset_left_child = root->left_child_offset;
    gint offset_right_grandchild = ((struct wg_tnode *)offsettoptr(db,offset_left_child))->right_child_offset;

    //first swtich: G goes to A's left child
    ee = (struct wg_tnode *)offsettoptr(db,offset_right_grandchild);
    root -> left_child_offset = ee -> right_child_offset;
    root -> left_subtree_height = ee -> right_subtree_height;
    if(ee -> right_child_offset != 0){
      ((struct wg_tnode *)offsettoptr(db,ee->right_child_offset))->parent_offset = ptrtooffset(db, root);
    }
    //second switch: F goes to B's right child
    bb = (struct wg_tnode *)offsettoptr(db,offset_left_child);
    bb -> right_child_offset = ee -> left_child_offset;
    bb -> right_subtree_height = ee -> left_subtree_height;
    if(ee -> left_child_offset != 0){
      ((struct wg_tnode *)offsettoptr(db,ee->left_child_offset))->parent_offset = offset_left_child;
    }
    //third switch: B goes to E's left child
    /* The Lehman/Carey "special" LR rotation - instead of creating
     * an internal node with one element, the values of what will become the
     * left child will be moved over to the parent, thus ensuring the internal
     * node is adequately filled. This is only allowed if E is a leaf.
     */
    if(ee->number_of_elements == 1 && !ee->right_child_offset &&\
      !ee->left_child_offset && bb->number_of_elements == WG_TNODE_ARRAY_SIZE){
      int i;

      /* Create space for elements from B */
      ee->array_of_values[bb->number_of_elements - 1] = ee->array_of_values[0];

      /* All the values moved are smaller than in E */
      for(i=1; i<bb->number_of_elements; i++)
        ee->array_of_values[i-1] = bb->array_of_values[i];
      ee->number_of_elements = bb->number_of_elements;

      /* Examine the new leftmost element to find current_min */
      ee->current_min = wg_get_field(db, (void *)offsettoptr(db,
        ee->array_of_values[0]), column);

      bb -> number_of_elements = 1;
      bb -> current_max = bb -> current_min;
    }

    //then switch the nodes
    ee -> left_child_offset = offset_left_child;
    ee -> left_subtree_height = max(bb->right_subtree_height,bb->left_subtree_height)+1;
    bb -> parent_offset = offset_right_grandchild;
    //fourth switch: A goes to E's right child
    ee -> right_child_offset = ptrtooffset(db, root);
    ee -> right_subtree_height = max(root->right_subtree_height,root->left_subtree_height)+1;
    root -> parent_offset = offset_right_grandchild;
    //for later grandparent fix
    r = ee;

  }else if(overw == RL_CASE){

/*               A                    E
*             C     B             A       B
*                 E   D  ->     C  G    F   D
*               G  F                    N
*                  N
*/
    struct wg_tnode *bb, *ee;
    //save some stuff into variables for later use
    gint offset_right_child = root->right_child_offset;
    gint offset_left_grandchild = ((struct wg_tnode *)offsettoptr(db,offset_right_child))->left_child_offset;

    //first swtich: G goes to A's left child
    ee = (struct wg_tnode *)offsettoptr(db,offset_left_grandchild);
    root -> right_child_offset = ee -> left_child_offset;
    root -> right_subtree_height = ee -> left_subtree_height;
    if(ee -> left_child_offset != 0){
      ((struct wg_tnode *)offsettoptr(db,ee->left_child_offset))->parent_offset = ptrtooffset(db, root);
    }

    //second switch: F goes to B's right child
    bb = (struct wg_tnode *)offsettoptr(db,offset_right_child);
    bb -> left_child_offset = ee -> right_child_offset;
    bb -> left_subtree_height = ee -> right_subtree_height;
    if(ee -> right_child_offset != 0){
      ((struct wg_tnode *)offsettoptr(db,ee->right_child_offset))->parent_offset = offset_right_child;
    }

    //third switch: B goes to E's right child
    /* "special" RL rotation - see comments for LR_CASE */
    if(ee->number_of_elements == 1 && !ee->right_child_offset &&\
      !ee->left_child_offset &&  bb->number_of_elements == WG_TNODE_ARRAY_SIZE){
      int i;

      /* All the values moved are larger than in E */
      for(i=1; i<bb->number_of_elements; i++)
        ee->array_of_values[i] = bb->array_of_values[i-1];
      ee->number_of_elements = bb->number_of_elements;

      /* Examine the new rightmost element to find current_max */
      ee->current_max = wg_get_field(db, (void *)offsettoptr(db,
        ee->array_of_values[ee->number_of_elements - 1]), column);

      /* Remaining B node array element should sit in slot 0 */
      bb->array_of_values[0] = \
        bb->array_of_values[bb->number_of_elements - 1];
      bb -> number_of_elements = 1;
      bb -> current_min = bb -> current_max;
    }

    ee -> right_child_offset = offset_right_child;
    ee -> right_subtree_height = max(bb->right_subtree_height,bb->left_subtree_height)+1;
    bb -> parent_offset = offset_left_grandchild;
    //fourth switch: A goes to E's right child

    ee -> left_child_offset = ptrtooffset(db, root);
    ee -> left_subtree_height = max(root->right_subtree_height,root->left_subtree_height)+1;
    root -> parent_offset = offset_left_grandchild;
    //for later grandparent fix
    r = ee;

  } else {
    /* catch an error case (can't really happen) */
    show_index_error(db, "tree rotate called with invalid argument, "\
      "index may have become corrupt");
    return -1;
  }

  //fix grandparent - regardless of current 'overweight' case

  if(grandparent == 0){//'grandparent' is index header data
    r->parent_offset = 0;
    //TODO more error check here
    TTREE_ROOT_NODE(hdr) = ptrtooffset(db,r);
  }else{//grandparent is usual node
    //printf("change grandparent node\n");
    r -> parent_offset = grandparent;
    if(g->left_child_offset == initialrootoffset){//new subtree must replace the left child of grandparent
      g->left_child_offset = ptrtooffset(db,r);
      g->left_subtree_height = max(r->left_subtree_height,r->right_subtree_height)+1;
    }else{
      g->right_child_offset = ptrtooffset(db,r);
      g->right_subtree_height = max(r->left_subtree_height,r->right_subtree_height)+1;
    }
  }

  return 0;
}

__device__ static gint ttree_remove_row(void *db, gint index_id, void * rec) {
  int i, found;
  gint key, rootoffset, column, boundtype, bnodeoffset;
  gint rowoffset;
  struct wg_tnode *node, *parent;
  wg_index_header *hdr = (wg_index_header *)offsettoptr(db,index_id);

  rootoffset = TTREE_ROOT_NODE(hdr);
#ifdef CHECK
  if(rootoffset == 0){
#ifdef WG_NO_ERRPRINT
#else
    //printf("index at offset %d does not exist\n", (int) index_id);
#endif
    return -1;
  }
#endif
  column = hdr->rec_field_index[0]; /* always one column for T-tree */
  key = wg_get_field(db, rec, column);
  rowoffset = ptrtooffset(db, rec);

  /* find bounding node for the value. Since non-unique values
   * are allowed, we will find the leftmost node and scan
   * right from there (we *need* the exact row offset).
   */

  bnodeoffset = wg_search_ttree_leftmost(db,
          rootoffset, key, &boundtype, NULL);
  node = (struct wg_tnode *)offsettoptr(db,bnodeoffset);

  //if bounding node does not exist - error
  if(boundtype != REALLY_BOUNDING_NODE) return -2;

  /* find the record inside the node. This is an expensive loop if there
   * are many repeated values, so unnecessary deleting should be avoided
   * on higher level.
   */
  found = -1;
  for(;;) {
    for(i=0;i<node->number_of_elements;i++){
      if(node->array_of_values[i] == rowoffset) {
        found = i;
        goto found_row;
      }
    }
    bnodeoffset = TNODE_SUCCESSOR(db, node);
    if(!bnodeoffset)
      break; /* no more successors */
    node = (struct wg_tnode *)offsettoptr(db,bnodeoffset);
    if(WG_COMPARE(db, node->current_min, key) == WG_GREATER)
      break; /* successor is not a bounding node */
  }

found_row:
  if(found == -1) return -3;

  //delete the key and rearrange other elements
  node->number_of_elements--;
  if(found < node->number_of_elements) { /* not the last element */
    /* slide the elements to the right of the found value
     * one step to the left */
    for(i=found; i<node->number_of_elements; i++)
      node->array_of_values[i] = node->array_of_values[i+1];
  }

  /* Update min/max */
  if(found==node->number_of_elements && node->number_of_elements != 0) {
    /* Rightmost element was removed, so new max should be updated to
     * the new rightmost value */
    node->current_max = wg_get_field(db, (void *)offsettoptr(db,
      node->array_of_values[node->number_of_elements - 1]), column);
  } else if(found==0 && node->number_of_elements != 0) {
    /* current_min removed, update to new leftmost value */
    node->current_min = wg_get_field(db, (void *)offsettoptr(db,
      node->array_of_values[0]), column);
  }

  //check underflow and take some actions if needed
  if(node->number_of_elements < 5){//TODO use macro
    //if the node is internal node - borrow its gratest lower bound from the node where it is
    if(node->left_child_offset != 0 && node->right_child_offset != 0){//internal node
#ifndef TTREE_CHAINED_NODES
      gint greatestlb = wg_ttree_find_glb_node(db,node->left_child_offset);
#else
      gint greatestlb = node->pred_offset;
#endif
      struct wg_tnode *glbnode = (struct wg_tnode *)offsettoptr(db, greatestlb);

      /* Make space for a new min value */
      for(i=node->number_of_elements; i>0; i--)
        node->array_of_values[i] = node->array_of_values[i-1];

      /* take the glb value (always the rightmost in the array) and
       * insert it in our node */
      node -> array_of_values[0] = \
        glbnode->array_of_values[glbnode->number_of_elements-1];
      node -> number_of_elements++;
      node -> current_min = glbnode -> current_max;
      if(node->number_of_elements == 1) /* we just got our first element */
        node->current_max = glbnode -> current_max;
      glbnode -> number_of_elements--;

      //reset new max for glbnode
      if(glbnode->number_of_elements != 0) {
        glbnode->current_max = wg_get_field(db, (void *)offsettoptr(db,
          glbnode->array_of_values[glbnode->number_of_elements - 1]), column);
      }

      node = glbnode;
    }
  }

  //now variable node points to the node which really lost an element
  //this is definitely leaf or half-leaf
  //if the node is empty - free it and rebalanc the tree
  parent = NULL;
  //delete the empty leaf
  if(node->left_child_offset == 0 && node->right_child_offset == 0 && node->number_of_elements == 0){
    if(node->parent_offset != 0){
      parent = (struct wg_tnode *)offsettoptr(db, node->parent_offset);
      //was it left or right child
      if(parent->left_child_offset == ptrtooffset(db,node)){
        parent->left_child_offset=0;
        parent->left_subtree_height=0;
      }else{
        parent->right_child_offset=0;
        parent->right_subtree_height=0;
      }
    }
#ifdef TTREE_CHAINED_NODES
    /* Remove the node from sequential chain */
    if(node->succ_offset) {
      struct wg_tnode *succ = \
        (struct wg_tnode *) offsettoptr(db, node->succ_offset);
      succ->pred_offset = node->pred_offset;
    } else {
      TTREE_MAX_NODE(hdr) = node->pred_offset;
    }
    if(node->pred_offset) {
      struct wg_tnode *pred = \
        (struct wg_tnode *) offsettoptr(db, node->pred_offset);
      pred->succ_offset = node->succ_offset;
    } else {
      TTREE_MIN_NODE(hdr) = node->succ_offset;
    }
#endif
    /* Free the node, unless it's the root node */
    if(node != offsettoptr(db, TTREE_ROOT_NODE(hdr))) {
      wg_free_tnode(db, ptrtooffset(db,node));
    } else {
      /* Set empty state of root node */
      node->current_max = WG_ILLEGAL;
      node->current_min = WG_ILLEGAL;
#ifdef TTREE_CHAINED_NODES
      TTREE_MAX_NODE(hdr) = TTREE_ROOT_NODE(hdr);
      TTREE_MIN_NODE(hdr) = TTREE_ROOT_NODE(hdr);
#endif
    }
    //rebalance if needed
  }

  //or if the node was a half-leaf, see if it can be merged with its leaf
  if((node->left_child_offset == 0 && node->right_child_offset != 0) || (node->left_child_offset != 0 && node->right_child_offset == 0)){
    int elements = node->number_of_elements;
    int left;
    struct wg_tnode *child;
    if(node->left_child_offset != 0){
      child = (struct wg_tnode *)offsettoptr(db, node->left_child_offset);
      left = 1;//true
    }else{
      child = (struct wg_tnode *)offsettoptr(db, node->right_child_offset);
      left = 0;//false
    }
    elements += child->number_of_elements;
    if(!(child->left_subtree_height == 0 && child->right_subtree_height == 0)){
      show_index_error(db,
        "index tree is not balanced, deleting algorithm doesn't work");
      return -4;
    }
    //if possible move all elements from child to node and free child
    if(elements <= WG_TNODE_ARRAY_SIZE){
      int i = node->number_of_elements;
      int j;
      node->number_of_elements = elements;
      if(left){
        /* Left child elements are all smaller than in current node */
        for(j=i-1; j>=0; j--){
          node->array_of_values[j + child->number_of_elements] = \
            node->array_of_values[j];
        }
        for(j=0;j<child->number_of_elements;j++){
          node->array_of_values[j]=child->array_of_values[j];
        }
        node->left_subtree_height=0;
        node->left_child_offset=0;
        node->current_min=child->current_min;
        if(!i) node->current_max=child->current_max; /* parent was empty */
      }else{
        /* Right child elements are all larger than in current node */
        for(j=0;j<child->number_of_elements;j++){
          node->array_of_values[i+j]=child->array_of_values[j];
        }
        node->right_subtree_height=0;
        node->right_child_offset=0;
        node->current_max=child->current_max;
        if(!i) node->current_min=child->current_min; /* parent was empty */
      }
#ifdef TTREE_CHAINED_NODES
      /* Remove the child from sequential chain */
      if(child->succ_offset) {
        struct wg_tnode *succ = \
          (struct wg_tnode *) offsettoptr(db, child->succ_offset);
        succ->pred_offset = child->pred_offset;
      } else {
        TTREE_MAX_NODE(hdr) = child->pred_offset;
      }
      if(child->pred_offset) {
        struct wg_tnode *pred = \
          (struct wg_tnode *) offsettoptr(db, child->pred_offset);
        pred->succ_offset = child->succ_offset;
      } else {
        TTREE_MIN_NODE(hdr) = child->succ_offset;
      }
#endif
      wg_free_tnode(db, ptrtooffset(db, child));
      if(node->parent_offset) {
        parent = (struct wg_tnode *)offsettoptr(db, node->parent_offset);
        if(parent->left_child_offset==ptrtooffset(db,node)){
          parent->left_subtree_height=1;
        }else{
          parent->right_subtree_height=1;
        }
      }
    }
  }

  //check balance and update subtree height data
  //stop when find a node where subtree heights dont change
  if(parent != NULL){
    int balance, height;
    for(;;) {
      balance = parent->left_subtree_height - parent->right_subtree_height;
      if(balance > 1 || balance < -1){//must rebalance
        //the current parent is root for balancing operation
        //rotarion fixes subtree heights in grandparent
        //determine the branch that causes overweight
        int overw = db_which_branch_causes_overweight(db,parent);
        //fix balance
        db_rotate_ttree(db,index_id,parent,overw);
      }
      else if(parent->parent_offset) {
        struct wg_tnode *gp;
        //manually set grandparent subtree heights
        height = max(parent->left_subtree_height,parent->right_subtree_height);
        gp = (struct wg_tnode *)offsettoptr(db, parent->parent_offset);
        if(gp->left_child_offset==ptrtooffset(db,parent)){
          gp->left_subtree_height=1+height;
        }else{
          gp->right_subtree_height=1+height;
        }
      }
      if(!parent->parent_offset)
        break; /* root node reached */
      parent = (struct wg_tnode *)offsettoptr(db, parent->parent_offset);
    }
  }
  return 0;
}

__device__
     static gint show_index_error(void* db, char* errmsg) {   // 2992
#ifdef WG_NO_ERRPRINT
#else
	// fprintf(stderr,"index error: %s\n",errmsg);
	printf("\nindex error: %s\n", errmsg);
#endif
	return -1;
}

__device__ static gint hash_remove_row(void *db, gint index_id, void *rec) {
  wg_index_header *hdr = (wg_index_header *)offsettoptr(db,index_id);
  gint i;
  gint values[MAX_INDEX_FIELDS];

  for(i=0; i<hdr->fields; i++) {
    values[i] = wg_get_field(db, rec, hdr->rec_field_index[i]);
  }
  return hash_recurse(db, hdr, NULL, 0, values, hdr->fields, rec,
    HASHIDX_OP_REMOVE, (hdr->type == WG_INDEX_TYPE_HASH_JSON));
}

__device__
     static gint show_index_error_nr(void* db, char* errmsg, gint nr) { // 3006
#ifdef WG_NO_ERRPRINT
#else
	// fprintf(stderr,"index error: %s %d\n", errmsg, (int) nr);
	printf("\nindex error: %s %d\n", errmsg, (int) nr);
#endif
	return -1;
}
